#include "hip/hip_runtime.h"
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#ifdef OUTPUT_CSV_FILE
#include <fstream>
#endif

#define MIN(a, b) ((a) < (b) ? (a) : (b))

#ifndef MSZ
#define MSZ (8 * 1024)
#endif
#ifndef KSZ
#define KSZ (2 * 1024)
#endif
#ifndef NSZ
#define NSZ 32
#endif
#ifndef SP
#define SP 0.3
#endif

#define VLC 32

/*
A = C B^T
A is a sparse matrix of size MSZ X KSZ
C is a dense matrix of size  MSZ X NSZ
B^T is a dense matrix of size NSZ x KSZ

*/

void Sddmm(const int *a_row_offsets,
           const int *a_column_indices, const float *c,
           const float *b, float *a_values)
{
    for (int i = 0; i < MSZ; ++i)
    {
        for (int j = a_row_offsets[i]; j < a_row_offsets[i + 1]; ++j)
        {
            int idx_n = a_column_indices[j];
            double accumulator = 0.0;
            for (int l = 0; l < NSZ; ++l)
            {
                accumulator += static_cast<double>(c[i * NSZ + l]) *
                               static_cast<double>(b[idx_n * NSZ + l]);
            }
            a_values[j] = static_cast<float>(accumulator);
        }
    }
}

double checksum(float *x, int n)
{
    double sum = 0.0;
    for (int i = 0; i < n; i++)
    {
        sum += x[i] * x[i];
    }
    sum = sqrt(sum / n);
    return sum;
}

void validate(float *x_gold, float *x, int n)
{
    double max_diff = 0.0;
    int maxi = 0;
    for (int i = 0; i < n; i++)
    {
        double diff = fabs(x_gold[i] - x[i]);
        if (diff > max_diff)
        {
            max_diff = diff;
            maxi = i;
        }
    }
    // print maxi and the maximum difference
    std::cout << "maxi = " << maxi << " max_diff = " << max_diff << std::endl;
    // print the values of the two arrays at maxi with 7 digits of precision
    std::cout.precision(7);
    std::cout << "gold output[" << maxi << "] = " << x_gold[maxi] << std::endl;
    std::cout << "computed[" << maxi << "] = " << x[maxi] << std::endl;
}

int main(int argc, char **argv)
{

    int nDevices;
    float peak_bw;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);

        peak_bw = 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6;
        printf("  Peak Memory Bandwidth (GB/s): %6.2f\n\n", peak_bw);
    }

    // create a random sparse matrix with size m*n and initialize it with random values using rand() function
    int mnnz = (1.01 * MSZ * KSZ * SP) + 1000;

    int *a_row_offsets = (int *)malloc((MSZ + 1) * sizeof(int));
    int *a_column_indices = (int *)malloc(mnnz * sizeof(int));
    // float *values = (float *)malloc(mnnz * sizeof(float));
    // initialize a_row_offsets and a_column_indices for a uniform random SP pattern
    a_row_offsets[0] = 0;
    for (int i = 0; i < MSZ; i++)
    {
        a_row_offsets[i + 1] = a_row_offsets[i];
        for (int j = 0; j < KSZ; j++)
        {
            if ((static_cast<float>(rand()) / static_cast<float>(RAND_MAX) < SP))
            {
                if (a_row_offsets[i + 1] >= mnnz)
                {
                    printf("nnz exceeding mnnz %d mnnz = %d\n", a_row_offsets[i + 1], mnnz);
                    exit(0);
                }

                a_column_indices[a_row_offsets[i + 1]] = j;
                a_row_offsets[i + 1]++;
            }
        }
    }

    int nnz = a_row_offsets[MSZ];
    printf("MSZ = %d KSZ = %d NSZ = %d SP = %f\n", MSZ, KSZ, NSZ, SP);
    printf("nnz = %d mnnz = %d\n", nnz, mnnz);

    float *c = (float *)malloc(MSZ * NSZ * sizeof(float));

    // create a random c array with size m*k and initialize it with random values using rand() function
    for (int i = 0; i < MSZ * NSZ; i++)
    {
        c[i] = (static_cast<float>(rand()) / static_cast<float>(RAND_MAX)) * 0.01;
    }
    // allocate rhs_matrix array of size n*k
    // we are interested in multiplication with rhs_matrix transpose,
    // so we will read columns of rhs_matrix stored as rows (continuous memory access)
    float *b = (float *)malloc(KSZ * NSZ * sizeof(float));
    // create a random b array with size n*k and initialize it with random values using rand() function
    for (int i = 0; i < KSZ * NSZ; i++)
    {
        b[i] = (static_cast<float>(rand()) / static_cast<float>(RAND_MAX)) * 0.01;
    }
    //
    float *a_values = (float *)malloc(nnz * sizeof(float));
    float *a_values_host = (float *)malloc(nnz * sizeof(float));
    //  initialize a_values with zeros using memset
    memset(a_values_host, 0, nnz * sizeof(float));
    Sddmm(a_row_offsets, a_column_indices, c, b, a_values_host);
    // print checksum for a_values
    

    // device arrays and copy data from host to device
    int *a_row_offsets_dev;
    hipMalloc((void **)&a_row_offsets_dev, (MSZ + 1) * sizeof(int));
    // copy data from host to device
    hipMemcpy(a_row_offsets_dev, a_row_offsets, (MSZ + 1) * sizeof(int), hipMemcpyHostToDevice);
    int *a_column_indices_dev;
    hipMalloc((void **)&a_column_indices_dev, nnz * sizeof(int));
    hipMemcpy(a_column_indices_dev, a_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    float *c_dev;
    hipMalloc((void **)&c_dev, MSZ * NSZ * sizeof(float));
    hipMemcpy(c_dev, c, MSZ * NSZ * sizeof(float), hipMemcpyHostToDevice);
    float *b_dev;
    hipMalloc((void **)&b_dev, KSZ * NSZ * sizeof(float));
    hipMemcpy(b_dev, b, KSZ * NSZ * sizeof(float), hipMemcpyHostToDevice);

    float *a_values_dev;
    hipMalloc((void **)&a_values_dev, nnz * sizeof(float));

    hipsparseHandle_t handle = NULL;
    hipsparseDnMatDescr_t matC, matB;
    hipsparseSpMatDescr_t matA;
    void *dBuffer = NULL;
    size_t bufferSize = 0;
    hipsparseCreate(&handle);

    float alpha = 1.0f;
    float beta = 0.0f;

    // Create dense matrix C
    hipsparseCreateDnMat(&matC, MSZ, NSZ, NSZ, c_dev,
                        HIP_R_32F, HIPSPARSE_ORDER_ROW);

    // Create dense matrix B
    hipsparseCreateDnMat(&matB, KSZ, NSZ, NSZ, b_dev,
                        HIP_R_32F, HIPSPARSE_ORDER_ROW);

    // Create sparse matrix C in CSR format
    hipsparseCreateCsr(&matA, MSZ, KSZ, nnz,
                      a_row_offsets_dev, a_column_indices_dev, a_values_dev,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

    // allocate an external buffer if needed
    hipsparseSDDMM_bufferSize(handle,
                             HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             HIPSPARSE_OPERATION_TRANSPOSE,
                             &alpha, matC, matB, &beta, matA, HIP_R_32F,
                             HIPSPARSE_SDDMM_ALG_DEFAULT, &bufferSize);

    // print buffer size
   // printf("sddmm_csr_example buffer size: %lld bytes\n", (long long)bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    hipEvent_t tstart, tstop;
    hipEventCreate(&tstart);
    hipEventCreate(&tstop);

    float timeKernel;
    float exectime = 123456789.0;

    for (int loop = 0; loop < 10; ++loop)
    {
        hipEventRecord(tstart);
        hipsparseSDDMM(handle,
                      HIPSPARSE_OPERATION_NON_TRANSPOSE,
                      HIPSPARSE_OPERATION_TRANSPOSE,
                      &alpha, matC, matB, &beta, matA, HIP_R_32F,
                      HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer);

        hipEventRecord(tstop);
        hipEventSynchronize(tstop);
        hipEventElapsedTime(&timeKernel, tstart, tstop);

        if (exectime > timeKernel)
            exectime = timeKernel;
    }

    printf("Kernel time in ms:  %.12f\n", exectime);
    double gflops1 = (static_cast<double>(nnz) * (2.0 * NSZ - 1.0)) / (exectime * 1.0E+6);
    
    printf("GFLOPS :  %8.2f\n", gflops1);    
    hipMemcpy(a_values, a_values_dev, nnz * sizeof(float), hipMemcpyDeviceToHost);
    printf("checksum_a_values from CPU = %f\n", checksum(a_values_host, nnz));
    printf("checksum_a_values from GPU = %f\n", checksum(a_values, nnz));
    validate(a_values_host, a_values, nnz);

    #ifdef OUTPUT_CSV_FILE
    std::ofstream csv_file;
    csv_file.open(OUTPUT_CSV_FILE, std::ios::app | std::ios::out);
    csv_file << MSZ << "," << KSZ << "," << NSZ << "," << SP << "," << gflops1 << std::endl;
    csv_file.close();
    #endif



    hipsparseDestroyDnMat(matC);
    hipsparseDestroyDnMat(matB);
    hipsparseDestroySpMat(matA);
    hipsparseDestroy(handle);
//free device memory
    hipFree(a_row_offsets_dev);
    hipFree(a_column_indices_dev);
    hipFree(c_dev);
    hipFree(b_dev);
    hipFree(a_values_dev);
    hipFree(dBuffer);
    



    // free host and device memory
    free(a_row_offsets);
    free(a_column_indices);
    free(a_values);
    free(c);
    free(b);

    return 0;
}
