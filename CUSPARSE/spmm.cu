#include "hip/hip_runtime.h"
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#ifdef OUTPUT_CSV_FILE
#include <fstream>
#endif

#define MIN(a, b) ((a) < (b) ? (a) : (b))

#ifndef MSZ
#define MSZ (8 * 1024)
#endif
#ifndef KSZ
#define KSZ (2 * 1024)
#endif
#ifndef NSZ
#define NSZ 32
#endif
#ifndef SP
#define SP 0.3
#endif

#define VLC 32

/*
C =  A B
A is a sparse matrix of size MSZ X KSZ
B is a dense matrix of size  KSZ x NSZ
C is a dense matrix of size  MSZ x NSZ

*/

void Spmm(const float *a_values,
          const int *a_row_offsets, const int *a_column_indices,
          const float *b, float *c)
{
    for (int i = 0; i < MSZ; ++i)
    {
        for (int j = 0; j < NSZ; ++j)
        {
            float accum = 0.0;
            for (int l = a_row_offsets[i]; l < a_row_offsets[i + 1]; ++l)
            {
                float a_val = a_values[l];
                // print a_val
                // std::cout << "a_val = " << a_val << std::endl;
                int a_col = a_column_indices[l];
                accum += a_val * b[a_col * NSZ + j];
            }
            c[i * NSZ + j] = accum;
        }
    }
}

double checksum(float *x, int n)
{
    double sum = 0.0;
    for (int i = 0; i < n; i++)
    {
        sum += x[i] * x[i];
    }
    sum = sqrt(sum / n);
    return sum;
}

void validate(float *x_gold, float *x, int n)
{
    double max_diff = 0.0;
    int maxi = 0;
    for (int i = 0; i < n; i++)
    {
        double diff = fabs(x_gold[i] - x[i]);
        if (diff > max_diff)
        {
            max_diff = diff;
            maxi = i;
        }
    }
    // print maxi and the maximum difference
    std::cout << "maxi = " << maxi << " max_diff = " << max_diff << std::endl;
    // print the values of the two arrays at maxi with 7 digits of precision
    std::cout.precision(7);
    std::cout << "gold output[" << maxi << "] = " << x_gold[maxi] << std::endl;
    std::cout << "computed[" << maxi << "] = " << x[maxi] << std::endl;
}

int main(int argc, char **argv)
{

    int nDevices;
    float peak_bw;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);

        peak_bw = 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6;
        printf("  Peak Memory Bandwidth (GB/s): %6.2f\n\n", peak_bw);
    }

    // create a random sparse matrix with size m*n and initialize it with random values using rand() function
    int mnnz = (1.01 * MSZ * KSZ * SP) + 1000;
    float *a_values = (float *)malloc(mnnz * sizeof(float));
    int *a_row_offsets = (int *)malloc((MSZ + 1) * sizeof(int));
    int *a_column_indices = (int *)malloc(mnnz * sizeof(int));
    // float *values = (float *)malloc(mnnz * sizeof(float));
    // initialize a_row_offsets and a_column_indices for a uniform random SP pattern
    a_row_offsets[0] = 0;
    for (int i = 0; i < MSZ; i++)
    {
        a_row_offsets[i + 1] = a_row_offsets[i];
        for (int j = 0; j < KSZ; j++)
        {
            if ((static_cast<float>(rand()) / static_cast<float>(RAND_MAX) < SP))
            {
                if (a_row_offsets[i + 1] >= mnnz)
                {
                    printf("nnz exceeding mnnz %d mnnz = %d\n", a_row_offsets[i + 1], mnnz);
                    exit(0);
                }

                a_column_indices[a_row_offsets[i + 1]] = j;
                a_values[a_row_offsets[i + 1]] = (static_cast<float>(rand()) / static_cast<float>(RAND_MAX)) * 0.01;
                a_row_offsets[i + 1]++;
            }
        }
    }

    int nnz = a_row_offsets[MSZ];
    printf("MSZ = %d KSZ = %d NSZ = %d SP = %f\n", MSZ, KSZ, NSZ, SP);
    printf("nnz = %d mnnz = %d\n", nnz, mnnz);

    float *b = (float *)malloc(KSZ * NSZ * sizeof(float));

    // create a random c array with size m*k and initialize it with random values using rand() function
    for (int i = 0; i < KSZ * NSZ; i++)
    {
        b[i] = (static_cast<float>(rand()) / static_cast<float>(RAND_MAX)) * 0.01;
    }
    float *c = (float *)malloc(MSZ * NSZ * sizeof(float));
    memset(c, 0, MSZ * NSZ * sizeof(float));

    Spmm(a_values, a_row_offsets, a_column_indices, b, c);
    // print checksum for a_values
    // printf("checksum_a_values = %f\n", checksum(c, MSZ * NSZ));

#if 1
    // device arrays and copy data from host to device
    int *a_row_offsets_dev;
    hipMalloc((void **)&a_row_offsets_dev, (MSZ + 1) * sizeof(int));
    // copy data from host to device
    hipMemcpy(a_row_offsets_dev, a_row_offsets, (MSZ + 1) * sizeof(int), hipMemcpyHostToDevice);
    int *a_column_indices_dev;
    hipMalloc((void **)&a_column_indices_dev, nnz * sizeof(int));
    hipMemcpy(a_column_indices_dev, a_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    float *a_values_dev;
    hipMalloc((void **)&a_values_dev, nnz * sizeof(float));
    hipMemcpy(a_values_dev, a_values, nnz * sizeof(float), hipMemcpyHostToDevice);

    float *b_dev;
    hipMalloc((void **)&b_dev, KSZ * NSZ * sizeof(float));
    hipMemcpy(b_dev, b, KSZ * NSZ * sizeof(float), hipMemcpyHostToDevice);

    float *c_dev;
    hipMalloc((void **)&c_dev, MSZ * NSZ * sizeof(float));
    hipMemset(c_dev, 0, MSZ * NSZ * sizeof(float));

    hipsparseHandle_t handle = NULL;
    hipsparseDnMatDescr_t matC, matB;
    hipsparseSpMatDescr_t matA;
    void *dBuffer = NULL;
    size_t bufferSize = 0;
    hipsparseCreate(&handle);

    float alpha = 1.0f;
    float beta = 0.0f;

    // Create dense matrix C
    hipsparseCreateDnMat(&matC, MSZ, NSZ, NSZ, c_dev,
                        HIP_R_32F, HIPSPARSE_ORDER_ROW);

    // Create dense matrix B
    hipsparseCreateDnMat(&matB, KSZ, NSZ, NSZ, b_dev,
                        HIP_R_32F, HIPSPARSE_ORDER_ROW);

    // Create sparse matrix C in CSR format
    hipsparseCreateCsr(&matA, MSZ, KSZ, nnz,
                      a_row_offsets_dev, a_column_indices_dev, a_values_dev,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

    // allocate an external buffer if needed
    hipsparseSpMM_bufferSize(
        handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, matB, &beta, matC, HIP_R_32F,
        HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize);

    // print buffer size
   // printf("sddmm_csr_example buffer size: %lld bytes\n", (long long)bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    hipEvent_t tstart, tstop;
    hipEventCreate(&tstart);
    hipEventCreate(&tstop);

    float timeKernel;
    float exectime = 123456789.0;

    for (int loop = 0; loop < 10; ++loop)
    {
        hipEventRecord(tstart);

        hipsparseSpMM(handle,
                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                     &alpha, matA, matB, &beta, matC, HIP_R_32F,
                     HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer);

        hipEventRecord(tstop);
        hipEventSynchronize(tstop);
        hipEventElapsedTime(&timeKernel, tstart, tstop);

        if (exectime > timeKernel)
            exectime = timeKernel;
    }

    printf("Kernel time in ms:  %.12f\n", exectime);
    float opcount = (2 * (float)nnz - MSZ) * NSZ;
    float gflops1 = (opcount) / (exectime * 1.0E+6);
    printf("GFLOPS:  %8.2f\n", gflops1);

    float *c_host;
    c_host = (float *)malloc(MSZ * NSZ * sizeof(float));
    hipMemcpy(c_host, c_dev, MSZ * NSZ * sizeof(float), hipMemcpyDeviceToHost);
    printf("checksum c values from CPU = %f\n", checksum(c, MSZ * NSZ));
    printf("checksum C values from GPU = %f\n", checksum(c_host, MSZ * NSZ));
    validate(c, c_host, MSZ * NSZ);

#ifdef OUTPUT_CSV_FILE
    std::ofstream csv_file;
    csv_file.open(OUTPUT_CSV_FILE, std::ios::app | std::ios::out);
    csv_file << MSZ << "," << KSZ << "," << NSZ << "," << SP << "," << gflops1 << std::endl;
    csv_file.close();
#endif

    hipsparseDestroyDnMat(matC);
    hipsparseDestroyDnMat(matB);
    hipsparseDestroySpMat(matA);
    hipsparseDestroy(handle);

#endif
    // free cuda memory
    hipFree(a_row_offsets_dev);
    hipFree(a_column_indices_dev);
    hipFree(a_values_dev);
    hipFree(b_dev);
    hipFree(c_dev);
    hipFree(dBuffer);

    // free host and device memory
    free(a_row_offsets);
    free(a_column_indices);
    free(a_values);
    free(c);
    free(b);

    return 0;
}
