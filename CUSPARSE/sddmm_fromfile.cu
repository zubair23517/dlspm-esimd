#include "hip/hip_runtime.h"
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#ifdef OUTPUT_CSV_FILE
#include <fstream>
#endif

#define MIN(a, b) ((a) < (b) ? (a) : (b))

/*
C =  A B
A is a sparse matrix of size MSZ X KSZ
B is a dense matrix of size  KSZ x NSZ
C is a dense matrix of size  MSZ x NSZ

*/

void Sddmm(const int *a_row_offsets,
           const int *a_column_indices, const float *c,
           const float *b, float *a_values, const int MSZ, const int NSZ)
{
    for (int i = 0; i < MSZ; ++i)
    {
        for (int j = a_row_offsets[i]; j < a_row_offsets[i + 1]; ++j)
        {
            int idx_n = a_column_indices[j];
            double accumulator = 0.0;
            for (int l = 0; l < NSZ; ++l)
            {
                accumulator += static_cast<double>(c[i * NSZ + l]) *
                               static_cast<double>(b[idx_n * NSZ + l]);
            }
            a_values[j] = static_cast<float>(accumulator);
        }
    }
}

double checksum(float *x, int n)
{
    double sum = 0.0;
    for (int i = 0; i < n; i++)
    {
        sum += x[i] * x[i];
    }
    sum = sqrt(sum / n);
    return sum;
}

double validate(float *x_gold, float *x, int n)
{
    double max_diff = 0.0;
    int maxi = 0;
    for (int i = 0; i < n; i++)
    {
        double diff = fabs(x_gold[i] - x[i]);
        if (diff > max_diff)
        {
            max_diff = diff;
            maxi = i;
        }
    }
    // print maxi and the maximum difference
    std::cout << "maxi = " << maxi << " max_diff = " << max_diff << std::endl;
    // print the values of the two arrays at maxi with 7 digits of precision
    std::cout.precision(7);
    std::cout << "gold output[" << maxi << "] = " << x_gold[maxi] << std::endl;
    std::cout << "computed[" << maxi << "] = " << x[maxi] << std::endl;

    return max_diff;
}

int main(int argc, char **argv)
{

    if (argc != 3)
        throw std::invalid_argument("Usage: ./app <filename>.smtx <batchsize>");

    std::string filename(argv[1]);
    const int NSZ = std::stoi(argv[2]);
    std::cout << "Filename = " << filename << ", batchsize (=NSZ) = " << NSZ << std::endl;

    int nDevices;
    float peak_bw;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);

        peak_bw = 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6;
        printf("  Peak Memory Bandwidth (GB/s): %6.2f\n\n", peak_bw);
    }

    // Load lhs matrix from file
    FILE *fp;
    fp = fopen(filename.c_str(), "r");
    if (fp == NULL)
    {
        throw std::logic_error("Error opening file!");
    }

    int MSZ, KSZ, nnz;
    if (fscanf(fp, "%d,%d,%d", &MSZ, &KSZ, &nnz) != 3)
        throw std::logic_error("Something went wrong readin M, K, nnz");

    printf("MSZ: %d, KSZ: %d, NSZ: %d, nnz: %d\n", MSZ, KSZ, NSZ, nnz);

    int *row_offsets = (int *)malloc((MSZ + 1) * sizeof(int));
    int *column_indices = (int *)malloc(nnz * sizeof(int));

    for (int i = 0; i < MSZ + 1; i++)
    {
        if (fscanf(fp, "%d", &row_offsets[i]) != 1)
            throw std::logic_error("Something went wrong reading row_offsets");
    }

    for (int i = 0; i < nnz; i++)
    {
        if (fscanf(fp, "%d", &column_indices[i]) != 1)
            throw std::logic_error("Something went wrong reading column_indices");
    }

    fclose(fp);

    float *c = (float *)malloc(MSZ * NSZ * sizeof(float));

    // create a random c array with size m*k and initialize it with random values using rand() function
    for (int i = 0; i < MSZ * NSZ; i++)
    {
        c[i] = (static_cast<float>(rand()) / static_cast<float>(RAND_MAX)) * 0.01;
    }
    // allocate rhs_matrix array of size n*k
    // we are interested in multiplication with rhs_matrix transpose,
    // so we will read columns of rhs_matrix stored as rows (continuous memory access)
    float *b = (float *)malloc(KSZ * NSZ * sizeof(float));
    // create a random b array with size n*k and initialize it with random values using rand() function
    for (int i = 0; i < KSZ * NSZ; i++)
    {
        b[i] = (static_cast<float>(rand()) / static_cast<float>(RAND_MAX)) * 0.01;
    }
    //
    float *a_values = (float *)malloc(nnz * sizeof(float));
    float *a_values_host = (float *)malloc(nnz * sizeof(float));
    //  initialize a_values with zeros using memset
    memset(a_values_host, 0, nnz * sizeof(float));
    Sddmm(row_offsets, column_indices, c, b, a_values_host, MSZ, NSZ);
    // print checksum for a_values

    // device arrays and copy data from host to device
    int *a_row_offsets_dev;
    hipMalloc((void **)&a_row_offsets_dev, (MSZ + 1) * sizeof(int));
    // copy data from host to device
    hipMemcpy(a_row_offsets_dev, row_offsets, (MSZ + 1) * sizeof(int), hipMemcpyHostToDevice);
    int *a_column_indices_dev;
    hipMalloc((void **)&a_column_indices_dev, nnz * sizeof(int));
    hipMemcpy(a_column_indices_dev, column_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    float *c_dev;
    hipMalloc((void **)&c_dev, MSZ * NSZ * sizeof(float));
    hipMemcpy(c_dev, c, MSZ * NSZ * sizeof(float), hipMemcpyHostToDevice);
    float *b_dev;
    hipMalloc((void **)&b_dev, KSZ * NSZ * sizeof(float));
    hipMemcpy(b_dev, b, KSZ * NSZ * sizeof(float), hipMemcpyHostToDevice);

    float *a_values_dev;
    hipMalloc((void **)&a_values_dev, nnz * sizeof(float));

    hipsparseHandle_t handle = NULL;
    hipsparseDnMatDescr_t matC, matB;
    hipsparseSpMatDescr_t matA;
    void *dBuffer = NULL;
    size_t bufferSize = 0;
    hipsparseCreate(&handle);

    float alpha = 1.0f;
    float beta = 0.0f;

    // Create dense matrix C
    hipsparseCreateDnMat(&matC, MSZ, NSZ, NSZ, c_dev,
                        HIP_R_32F, HIPSPARSE_ORDER_ROW);

    // Create dense matrix B
    hipsparseCreateDnMat(&matB, KSZ, NSZ, NSZ, b_dev,
                        HIP_R_32F, HIPSPARSE_ORDER_ROW);

    // Create sparse matrix C in CSR format
    hipsparseCreateCsr(&matA, MSZ, KSZ, nnz,
                      a_row_offsets_dev, a_column_indices_dev, a_values_dev,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

    // allocate an external buffer if needed
    hipsparseSDDMM_bufferSize(handle,
                             HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             HIPSPARSE_OPERATION_TRANSPOSE,
                             &alpha, matC, matB, &beta, matA, HIP_R_32F,
                             HIPSPARSE_SDDMM_ALG_DEFAULT, &bufferSize);

    // print buffer size
    // printf("sddmm_csr_example buffer size: %lld bytes\n", (long long)bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    hipEvent_t tstart, tstop;
    hipEventCreate(&tstart);
    hipEventCreate(&tstop);

    float timeKernel;
    float exectime = 123456789.0;

    for (int loop = 0; loop < 10; ++loop)
    {
        hipEventRecord(tstart);
        hipsparseSDDMM(handle,
                      HIPSPARSE_OPERATION_NON_TRANSPOSE,
                      HIPSPARSE_OPERATION_TRANSPOSE,
                      &alpha, matC, matB, &beta, matA, HIP_R_32F,
                      HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer);

        hipEventRecord(tstop);
        hipEventSynchronize(tstop);
        hipEventElapsedTime(&timeKernel, tstart, tstop);

        if (exectime > timeKernel)
            exectime = timeKernel;
    }

    printf("Kernel time in ms:  %.12f\n", exectime);
    double gflops1 = (static_cast<double>(nnz) * (2.0 * NSZ - 1.0)) / (exectime * 1.0E+6);

    printf("GFLOPS :  %8.2f\n", gflops1);
    hipMemcpy(a_values, a_values_dev, nnz * sizeof(float), hipMemcpyDeviceToHost);
    printf("checksum_a_values from CPU = %f\n", checksum(a_values_host, nnz));
    printf("checksum_a_values from GPU = %f\n", checksum(a_values, nnz));
    const double max_diff = validate(a_values_host, a_values, nnz);

#ifdef OUTPUT_CSV_FILE
    std::ofstream csv_file;
    csv_file.open(OUTPUT_CSV_FILE, std::ios::app | std::ios::out);
    csv_file << MSZ << "," << KSZ << "," << NSZ << "," << (double)nnz / (MSZ * KSZ) << "," << gflops1 << "," << max_diff << std::endl;
    csv_file.close();
#endif

    hipsparseDestroyDnMat(matC);
    hipsparseDestroyDnMat(matB);
    hipsparseDestroySpMat(matA);
    hipsparseDestroy(handle);
    // free device memory
    hipFree(a_row_offsets_dev);
    hipFree(a_column_indices_dev);
    hipFree(c_dev);
    hipFree(b_dev);
    hipFree(a_values_dev);
    hipFree(dBuffer);

    // free host and device memory
    free(row_offsets);
    free(column_indices);
    free(a_values);
    free(c);
    free(b);

    return 0;
}